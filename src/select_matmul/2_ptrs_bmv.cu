
#include <hip/hip_runtime.h>
#include "hipblas.h"

__global__ void ptrs(const int32_t size, const int32_t stride, const int32_t* s, float* base_ptr, float* *out)
{
    const int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        const int32_t offset = (s == nullptr) ? tid : s[tid];
        out[tid] = base_ptr + stride * offset;
    }
}

void PtrsThenBMV(
    hipblasHandle_t handle,
    const int32_t batch_size,
    const int32_t input_size,
    const int32_t output_size,
    const int32_t buckets,
    float* A,
    float* x,
    int32_t* s,
    float* y,
    void* intmdt)
{
    // controls the layout of matrix A
    // if true then result should match Matmul + Select
    // if false then results will not match
    const bool interleave = true;

    float** As = (float**) intmdt;
    float** xs = As + batch_size;
    float** ys = xs + batch_size;

    const int32_t stride = (interleave) ? output_size : output_size * buckets;
    const int32_t ldA = (interleave) ? output_size * buckets : output_size;
    const float alpha = 1.0F;
    const float beta = 0.0F;

    const size_t threads = 512;
    const size_t blocks = (batch_size + threads - 1) / threads;

    ptrs<<<blocks, threads>>>(batch_size, stride, s, A, As);
    ptrs<<<blocks, threads>>>(batch_size, input_size, nullptr, x, xs);
    ptrs<<<blocks, threads>>>(batch_size, output_size, nullptr, y, ys);

    hipblasSgemvBatched(
        handle,
        HIPBLAS_OP_N,
        output_size, input_size,
        &alpha,
        As, ldA,
        xs, 1,
        &beta,
        ys, 1,
        batch_size
    );
}
