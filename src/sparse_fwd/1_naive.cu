
#include <hip/hip_runtime.h>
__global__ void sparse_fwd_naive_kernel(const int32_t m, const int32_t nnz, const float* A, const int32_t* x, float* y)
{
    const int32_t row = threadIdx.x + blockDim.x * blockIdx.x;
    const int32_t* tx = x + nnz * blockIdx.y;
    float* ty = y + m * blockIdx.y; 

    if (row < m)
    {
        float sum = 0;
        for (int i = 0; i < nnz; i += 1)
        {
            const int j = tx[i];
            if (j != -1)
            {
                sum += A[m * j + row];
            }
        }

        ty[row] = sum;
    }
}
